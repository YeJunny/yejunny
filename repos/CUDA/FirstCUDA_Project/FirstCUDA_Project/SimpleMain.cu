#include "hip/hip_runtime.h"
#include <iostream>
//#include "hip/hip_runtime.h"

using namespace std;

__global__ void AddNumber(int* dest, int* source)
{
	*dest += *source;
}

int main() 
{
	int number1 = 5;
	int number2 = 9;

	int* destNumber1;
	int* destNumber2;

	if (hipMalloc(&destNumber1, sizeof(int)) == hipSuccess)
	{
		printf("Success\n");
	}
	if (hipMalloc(&destNumber2, sizeof(int)) == hipSuccess)
	{
		printf("Success\n");
	}

	hipMemcpy(destNumber1, &number1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(destNumber2, &number2, sizeof(int), hipMemcpyHostToDevice);

	AddNumber << <1, 1 >> > (destNumber1, destNumber2);

	hipMemcpy(&number1, destNumber1, sizeof(int), hipMemcpyDeviceToHost);

	cout << "Result AddNumber() : " << number1 << endl;

	hipFree(destNumber1);
	hipFree(destNumber2);

	return 0;
}